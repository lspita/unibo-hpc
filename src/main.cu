
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

__global__ void ker() {
  const uint32_t id = threadIdx.x;
  printf("Thread ID: %u\n", id);
}

int main(void) {
  printf("Hello, world!\n");
  ker<<<1, 1>>>();

  const hipError_t err = hipDeviceSynchronize();
  if (err == hipSuccess) {
    printf("CUDA: success\n");
  } else {
    const int32_t _ =
        fprintf(stderr, "CUDA: error: %s\n", hipGetErrorString(err));
    return 1;
  }

  return 0;
}
