/****************************************************************************
 *
 * cuda-multigpu.cu - CUDA multi GPU example
 *
 * Copyright (C) 2024 Moreno Marzolla
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define BLKDIM 1024

/* compute the sum of the content of v[BLKDIM] and store the result in v[0] */
__global__
void reduce( int *v )
{
    __shared__ int temp[BLKDIM];
    int lindex = threadIdx.x;
    int bsize = blockDim.x / 2;

    temp[lindex] = v[lindex];

    /* wait for all threads to finish the copy operation */
    __syncthreads();

    /* All threads within the block cooperate to compute the local sum */
    while ( bsize > 0 ) {
        if ( lindex < bsize ) {
            temp[lindex] += temp[lindex + bsize];
        }
        bsize = bsize / 2;
        /* threads must synchronize before performing the next
           reduction step */
        __syncthreads();
    }

    if ( 0 == lindex ) {
        v[0] = temp[0];
    }
}

void init(int *v, int len, int val)
{
    for (int i=0; i<len; i++) {
        v[i] = val;
    }
}

int main( void )
{
    int *p0, *d_p0, *p1, *d_p1;
    int r0, r1;

    const size_t size = BLKDIM * sizeof(*p0);

    hipHostMalloc( &p0, size , hipHostMallocDefault); init(p0, BLKDIM, 1);
    hipHostMalloc( &p1, size , hipHostMallocDefault); init(p1, BLKDIM, 2);

    hipSetDevice(0);            // Set device 0 as current
    hipMalloc(&d_p0, size);     // Allocate memory on device 0
    hipMemcpyAsync(d_p0, p0, size, hipMemcpyHostToDevice);
    reduce<<<1, BLKDIM>>>(d_p0); // Launch kernel on device 0
    hipMemcpyAsync(&r0, d_p0, sizeof(r0), hipMemcpyDeviceToHost);

    hipSetDevice(1);            // Set device 1 as current
    hipMalloc(&d_p1, size);     // Allocate memory on device 1
    hipMemcpyAsync(d_p1, p1, size, hipMemcpyHostToDevice);
    reduce<<<1, BLKDIM>>>(d_p1); // Launch kernel on device 1
    hipMemcpyAsync(&r1, d_p1, sizeof(r1), hipMemcpyDeviceToHost);

    hipSetDevice(0);
    hipDeviceSynchronize();
    hipFree(d_p0);
    // the CPU can use r0 here
    printf("r0 = %d\n", r0); assert(r0 == BLKDIM);

    hipSetDevice(1);
    hipDeviceSynchronize();
    hipFree(d_p1);
    // the CPU can use r1 here
    printf("r1 = %d\n", r1); assert(r1 == 2*BLKDIM);

    hipHostFree(p0);
    hipHostFree(p1);

    return 0;
}
